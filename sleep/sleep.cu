#include<unistd.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>
#include<chrono>
#include<hip/hip_runtime_api.h>
#include<cstdio>
#include<cstdint>

using namespace std;
using namespace chrono;

__device__ void gpu_sleep_0_bak(volatile clock_t* d_o, clock_t clock_count){
    clock_t start = clock();
    clock_t now;
    clock_t cycles;

    for(;;){
        now = clock();
        cycles = now > start ? now - start : now + (0xffffffff - start);
        if (cycles >= clock_count){
            break;
        }
    }

    *d_o = cycles;
}

__global__ void gpu_sleep_0(clock_t clock_count){
    static volatile clock_t buffer;
    gpu_sleep_0_bak(&buffer, clock_count);
}

__device__ void gpu_sleep_1_bak(unsigned long sleep_cycles)
{
    unsigned long start = clock64();
    unsigned long cycles_elapsed;
    do { cycles_elapsed = clock64() - start; } 
    while (cycles_elapsed < sleep_cycles);
}

__global__ void gpu_sleep_1(clock_t clock_count){
    gpu_sleep_1_bak(clock_count);
}


/*
__device__ void gpu_sleep_2_bak(unsigned int sleep_cycles)
{
    __nanosleep(sleep_cycles);
}

__global__ void gpu_sleep_2(unsigned int clock_count){

    //clock_count += 1;
    __nanosleep(clock_count);
    //gpu_sleep_2_bak(clock_count);
}
*/

void gpu_sleeper(const int device, const unsigned long t, intptr_t stream_ptr){
    hipStream_t stream = reinterpret_cast<hipStream_t>(stream_ptr);
    gpu_sleep_1<<<1, 1, device, stream>>>(t);
}

/*
int main(int argc, char* argv[]){

    int t = atoi(argv[1]);

    cudaStream_t stream;
    cudaStreamCreate(&stream);

    auto time_start = chrono::high_resolution_clock::now();
    gpu_sleeper(t, stream);
    cudaStreamSynchronize(stream);
    auto time_stop = chrono::high_resolution_clock::now();
    printf("Time elapsed: %f \n", chrono::duration<double>(time_stop - time_start)); 


    time_start = chrono::high_resolution_clock::now();
    gpu_sleeper(t, stream);
    cudaStreamSynchronize(stream);
    time_stop = chrono::high_resolution_clock::now();
    printf("Time elapsed: %f \n", chrono::duration<double>(time_stop - time_start)); 

}
*/
